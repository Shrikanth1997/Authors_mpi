
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

struct node{
	int dst;
	struct node* next;
};

struct list{
	struct node *head;
};

struct graph{
	int n;
	struct list* set;
};

struct node* new_node(int dst){
	struct node* newnode = (struct node*)malloc(sizeof(struct node));
	newnode -> dst = dst;
	newnode -> next = NULL;

	return newnode;
}

struct graph* new_graph(int n){
	struct graph* newgraph = (struct graph*)malloc(sizeof(struct node));
	newgraph -> n = n;
	
	newgraph -> set = (struct list*)malloc(n * sizeof(struct list)) ;

	int i;
	for(i=0;i<n;i++)
		newgraph->set[i].head = NULL;

	return newgraph;

}

void addEdge(struct graph* gph, int src, int dst){
	struct node* newnode = new_node(dst);
	newnode->next = gph->set[src].head;
	gph->set[src].head = newnode;

	newnode = new_node(src);
        newnode->next = gph->set[dst].head;
        gph->set[dst].head = newnode;
}


__global__ void add( int *a, int *b, int *c, int vector_size ) {
    
    // Calculate the index in the vector for the thread using the internal variables
    int tid = blockIdx.x * blockDim.x + threadIdx.x; // HERE
    
    // This if statement is added in case we have more threads executing
    // Than number of elements in the vectors. How can this help?
    if (tid < vector_size){
        
        // Compute the addition
        c[tid] = a[tid] + b[tid];
        
    }
}

long get_vert(char *str){
	char vert[20];
	int space_count = 0;
	int num_vert=0;	
	
	int i=0, j=0;
	while(str[i] != '\n'){
	
		if(str[i] == ' ')
			space_count++;
		if(space_count == 2){
			vert[j] = str[i];
			j++;
		}
		else if(space_count>2)	
			break;
		i++;
	}
	vert[j] = '\0';
    	//printf("%s\n", vert);
	num_vert = atoi(vert);
    	//printf("%d\n", num_vert);
	return num_vert;
	
}

int get_src(char *str){
	char s[20];
        int space_count = 0;
        int src=0;

        int i=0, j=0;
        while(str[i] != '\n'){

                if(str[i] == ' ')
                        space_count++;
                if(space_count == 0){
                        s[j] = str[i];
                        j++;
                }
		else
			break;
                i++;
        }
        s[j] = '\0';
        //printf("%s\n", s);
        src = atoi(s);
        //printf("%d\n", src);
        return src;
}

int get_dst(char *str){
	char d[20];
        int space_count = 0;
        int dst=0;

        int i=0, j=0;
        while(str[i] != '\n'){

                if(str[i] == ' ')
                        space_count++;
                if(space_count == 1){
                        d[j] = str[i];
                        j++;
                }
		else if(space_count>1)
			break;
                i++;
        }
        d[j] = '\0';
        //printf("%s\n", d);
        dst = atoi(d);
        //printf("%d\n", dst);
        return dst;
}

int compare (const void * a, const void * b)
{
  return ( *(int*)b - *(int*)a );
}


int main( int argc, char* argv[] ) { 

    // Parse Input arguments

    // Check the number of arguments
    if (argc != 3) {
        // Tell the user how to run the program
        printf ("Usage: %s vector_size block_size\n", argv[0]);
        // "Usage messages" are a conventional way of telling the user
        // how to run a program if they enter the command incorrectly.
        return 1;
    }
    
    // Set GPU Variables based on input arguments
    int vector_size = atoi(argv[1]);
    int block_size  = atoi(argv[2]);
    int grid_size   = ((vector_size-1)/block_size) + 1;

    // Set device that we will use for our cuda code
    hipSetDevice(0);
        
    // Time Variables
    hipEvent_t start, stop;
    float time;
    hipEventCreate (&start);
    hipEventCreate (&stop);

    // Input Arrays and variables
    int *a        = new int [vector_size]; 
    int *b        = new int [vector_size]; 
    int *c_cpu    = new int [vector_size]; 
    int *c_gpu    = new int [vector_size];

    // Pointers in GPU memory
    int *dev_a;
    int *dev_b;
    int *dev_c;

    // fill the arrays 'a' and 'b' on the CPU
    printf("Initializing input arrays.\n");
    for (int i = 0; i < vector_size; i++) {
        a[i] = rand()%10;
        b[i] = rand()%10;
    }

    //
    // CPU Calculation
    //////////////////

    printf("Running sequential job.\n");
    hipEventRecord(start,0);

    // Calculate C in the CPU
    for (int i = 0; i < vector_size; i++) {
            c_cpu[i] = a[i] + b[i];
    }

    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    printf("\tSequential Job Time: %.2f ms\n", time);

    int actual_size = vector_size * sizeof(int);

    // allocate the memory on the GPU
    hipMalloc(&dev_a,actual_size);
    hipMalloc(&dev_b,actual_size);
    hipMalloc(&dev_c,actual_size);

    // copy the arrays 'a' and 'b' to the GPU
    hipMemcpy(dev_a,a,actual_size,hipMemcpyHostToDevice);
    hipMemcpy(dev_b,b,actual_size,hipMemcpyHostToDevice);

    //
    // GPU Calculation
    ////////////////////////

    printf("Running parallel job.\n");

    hipEventRecord(start,0);

    // call the kernel
    //add<<<grid_size,block_size>>>(dev_a,dev_b,dev_c,actual_size);
    add<<<vector_size,1>>>(dev_a,dev_b,dev_c,actual_size);
    
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&time, start, stop);
    printf("\tParallel Job Time: %.2f ms\n", time);

    // copy the array 'c' back from the GPU to the CPU
    // HERE (there's one more at the end, don't miss it!)
    hipMemcpy(c_gpu,dev_c,actual_size,hipMemcpyDeviceToHost);
    
    // compare the results
    int error = 0;
    for (int i = 0; i < vector_size; i++) {
        if (c_cpu[i] != c_gpu[i]){
            error = 1;
            printf( "Error starting element %d, %d != %d\n", i, c_gpu[i], c_cpu[i] );    
        }
        if (error) break; 
    }

    if (error == 0){
        printf ("Correct result. No errors were found.\n");
    }

    // free CPU data
    free (a);
    free (b);
    free (c_cpu);
    free (c_gpu);

    // free the memory allocated on the GPU
    // HERE
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    return 0;
}

